#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "grad_in_gpu.cuh"

#define gt_col(y, x) gt_col[(y) * width + (x)]

__global__ void grad_update_for_one_thread(vertex *vertices, int size, float learning_rate) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // printf("idx: %d, col: %f, grad_col: %f\n", idx, vertices[idx].col, vertices[idx].grad_col);
        vertices[idx].col -= learning_rate * vertices[idx].grad_col;
        vertices[idx].col = min(1.0f, max(0.0f, vertices[idx].col));
        vertices[idx].grad_col = 0;
    }
}

// calculate the gradient of the color of the vertex
__global__ void grad_backwards_for_one_thread(vertex *vertices, Node *nodes, float *gt_col, int width, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int ori_y = int(idx / width);
        int ori_x = idx % width;
        float y = int(idx / width) + 0.5;
        float x = idx % width + 0.5;

        // find the node that contains the pixel
        Node node = nodes[0];
        int node_index = 0;
        while (node.node_index[0] != -1) {  // if the node has subnodes

            float mid_x = (vertices[node.vertex_index[0]].x + vertices[node.vertex_index[2]].x) / 2.0;  // (x0 + x2) / 2
            float mid_y = (vertices[node.vertex_index[0]].y + vertices[node.vertex_index[2]].y) / 2.0;  // (y0 + y2) / 2

            if (y >= mid_y) {
                if (x >= mid_x) {  // 0-phase
                    node_index = node.node_index[0];
                    node = nodes[node.node_index[0]];
                } else {  // 1-phase
                    node_index = node.node_index[1];
                    node = nodes[node.node_index[1]];
                }
            } else {
                if (x >= mid_x) {  // 3-phase
                    node_index = node.node_index[3];
                    node = nodes[node.node_index[3]];
                } else {  // 2-phase
                    node_index = node.node_index[2];
                    node = nodes[node.node_index[2]];
                }
            }
        }

        // get the four vertices of the node
        int *vertex_idx = node.vertex_index;
        vertex rt = vertices[vertex_idx[0]];
        vertex lt = vertices[vertex_idx[1]];
        vertex lb = vertices[vertex_idx[2]];
        vertex rb = vertices[vertex_idx[3]];

        // bilinear interpolation
        float right = rt.x;
        float top = rt.y;
        float left = lb.x;
        float bottom = lb.y;

        float col_rt = rt.col;
        float col_lt = lt.col;
        float col_lb = lb.col;
        float col_rb = rb.col;

        float sum_inv = 1 / ((right - left) * (top - bottom));
        float w_rt = (x - left) * (y - bottom) * sum_inv;
        float w_lt = (right - x) * (y - bottom) * sum_inv;
        float w_lb = (right - x) * (top - y) * sum_inv;
        float w_rb = (x - left) * (top - y) * sum_inv;

        float w = w_lb * col_lb + w_lt * col_lt + w_rb * col_rb + w_rt * col_rt;

        // calculate the loss
        float c = gt_col(ori_y, ori_x);
        float grad = 2 * (w - c);
        float loss = grad * grad / 4;

        // if (x == 0 && y == 0)
        // printf("x: %d, y: %d, w: %f, c: %f, grad: %f, loss: %f\n", x, y, w, c, grad, loss);

        // calculate the gradient of the color of the four vertices
        atomicAdd(&vertices[vertex_idx[0]].grad_col, grad * w_rt);
        atomicAdd(&vertices[vertex_idx[1]].grad_col, grad * w_lt);
        atomicAdd(&vertices[vertex_idx[2]].grad_col, grad * w_lb);
        atomicAdd(&vertices[vertex_idx[3]].grad_col, grad * w_rb);

        // calculate the loss of the node
        atomicAdd(&nodes[node_index].loss_sum, loss);
    }
}

void grad_in_gpu(vertex *vertices,
                 Node *nodes,
                 float *gt_col,
                 int vertice_size,
                 int node_size,
                 int gt_col_size,
                 int width,
                 int pixel_size,
                 float learning_rate) {
    // std::cout << "vertices grad_col before: " << std::endl;
    // for (int i = 0; i < vertice_size; i++) {
    //     std::cout << vertices[i].col << ' ' << vertices[i].grad_col << std::endl;
    // }

    // std::cout << "nodes loss_sum before: " << std::endl;
    // for (int i = 0; i < node_size; i++) {
    //     std::cout << nodes[i].loss_sum << std::endl;
    // }
    // vertices data copy to GPU
    vertex *d_vertices;
    hipMalloc(&d_vertices, vertice_size * sizeof(vertex));
    hipMemcpy(d_vertices, vertices, vertice_size * sizeof(vertex), hipMemcpyHostToDevice);

    // nodes data copy to GPU
    Node *d_nodes;
    hipMalloc(&d_nodes, node_size * sizeof(Node));
    hipMemcpy(d_nodes, nodes, node_size * sizeof(Node), hipMemcpyHostToDevice);

    // gt_col data copy to GPU
    float *d_gt_col;
    hipMalloc(&d_gt_col, gt_col_size * sizeof(float));
    hipMemcpy(d_gt_col, gt_col, gt_col_size * sizeof(float), hipMemcpyHostToDevice);

    // grad_backwards
    int back_block_size = 256;
    int back_grid_size = (pixel_size + back_block_size - 1) / back_block_size;
    grad_backwards_for_one_thread<<<back_grid_size, back_block_size>>>(d_vertices, d_nodes, d_gt_col, width, pixel_size);

    hipDeviceSynchronize();

    // update
    int update_block_size = 256;
    int update_grid_size = (vertice_size + update_block_size - 1) / update_block_size;
    grad_update_for_one_thread<<<update_grid_size, update_block_size>>>(d_vertices, vertice_size, learning_rate);
    hipDeviceSynchronize();

    // vertices data copy back to CPU
    hipMemcpy(vertices, d_vertices, vertice_size * sizeof(vertex), hipMemcpyDeviceToHost);
    hipFree(d_vertices);

    // nodes data copy back to CPU
    hipMemcpy(nodes, d_nodes, node_size * sizeof(Node), hipMemcpyDeviceToHost);
    hipFree(d_nodes);

    // gt_col data copy back to CPU
    hipMemcpy(gt_col, d_gt_col, gt_col_size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_gt_col);

    // for (int i = 0; i < node_size; i++) {
    //     std::cout << nodes[i].loss_sum << std::endl;
    // }
}
